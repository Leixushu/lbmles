#include "hip/hip_runtime.h"
/*

Simulation of flow inside a 2D square cavity
using the lattice Boltzmann method (LBM)

Written by:       Abhijit Joshi (abhijit@accelereyes.com)

Last modified on: Thursday, July 18 2013 @12:08 pm

Build instructions: make (uses Makefile present in this folder)

Run instructions: optirun ./gpu_lbm

*/

#include<iostream>
#include<stdio.h>
#include<arrayfire.h>
using namespace af;

// problem parameters

const int     N = 128;                  // number of node points along X and Y (cavity length in lattice units)
const int     TIME_STEPS = 1000000;     // number of time steps for which the simulation is run
const double  REYNOLDS_NUMBER = 1E4;    // REYNOLDS_NUMBER = LID_VELOCITY * N / kinematicViscosity

// don't change these unless you know what you are doing

const int     Q = 9;                    // number of discrete velocity aections used
const double  DENSITY = 2.7;            // fluid density in lattice units
const double  LID_VELOCITY = 0.05;      // lid velocity in lattice units

// initialize values for aection vectors, density, velocity and distribution functions on the GPU

__global__ void initialize(const int N, const int Q, const double DENSITY, const double LID_VELOCITY, 
                           double *ex, double *ey, double *wt, int *oppos, 
                           double *rho, double *ux, double *uy, double* sigma, 
                           double *f, double *feq, double *f_new)
{
    // compute the global "i" and "j" location handled by this thread

    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    // bound checking
    if( (i > (N-1)) || (j > (N-1)) ) return;

    // D2Q9 model base velocities and weights

    ex[0] =  0.0;   ey[0] =  0.0;   wt[0] = 4.0 /  9.0;
    ex[1] =  1.0;   ey[1] =  0.0;   wt[1] = 1.0 /  9.0;
    ex[2] =  0.0;   ey[2] =  1.0;   wt[2] = 1.0 /  9.0;
    ex[3] = -1.0;   ey[3] =  0.0;   wt[3] = 1.0 /  9.0;
    ex[4] =  0.0;   ey[4] = -1.0;   wt[4] = 1.0 /  9.0;
    ex[5] =  1.0;   ey[5] =  1.0;   wt[5] = 1.0 / 36.0;
    ex[6] = -1.0;   ey[6] =  1.0;   wt[6] = 1.0 / 36.0;
    ex[7] = -1.0;   ey[7] = -1.0;   wt[7] = 1.0 / 36.0;
    ex[8] =  1.0;   ey[8] = -1.0;   wt[8] = 1.0 / 36.0;

    // define opposite (anti) aections (useful for implementing bounce back)

    oppos[0] = 0;      //      6        2        5
    oppos[1] = 3;      //               ^
    oppos[2] = 4;      //               |
    oppos[3] = 1;      //               |
    oppos[4] = 2;      //      3 <----- 0 -----> 1
    oppos[5] = 7;      //               |
    oppos[6] = 8;      //               |
    oppos[7] = 5;      //               v
    oppos[8] = 6;      //      7        4        8

    // natural index for location (i,j)

    const int index = i*N+j;  // column-ordering

    // initialize density and velocity fields inside the cavity

      rho[index] = DENSITY;
       ux[index] = 0.0;
       uy[index] = 0.0;
    sigma[index] = 0.0;

    // specify boundary condition for the moving lid

    if(j==0) ux[index] = LID_VELOCITY;

    // assign initial values for distribution functions
    // along various aections using equilibriu, functions

    for(int a=0;a<Q;a++) {

        int index_f = a + index*Q;

        double edotu = ex[a]*ux[index] + ey[a]*uy[index];
        double udotu = ux[index]*ux[index] + uy[index]*uy[index];

        feq[index_f]   = rho[index] * wt[a] * (1.0 + 3.0*edotu + 4.5*edotu*edotu - 1.5*udotu);
        f[index_f]     = feq[index_f];
        f_new[index_f] = feq[index_f];

    }
}

// this function updates the values of the distribution functions at all points along all aections
// carries out one lattice time-step (streaming + collision) in the algorithm

__global__ void collideAndStream( // READ-ONLY parameters (used by this function but not changed)
                                 const int N, const int Q, const double DENSITY, const double LID_VELOCITY, const double REYNOLDS_NUMBER,
                                 const double *ex,      // x-component of aection vector
                                 const double *ey,      // x-component of aection vector
                                 const double *wt,   // weight factor for each aection
                                 const int *oppos,        // anti (opposite) vector for each aection

                                 // READ + WRITE parameters (get updated in this function)

                                 double *rho,         // density
                                 double *ux,         // X-velocity
                                 double *uy,         // Y-velocity
                                 double *sigma,      // rate-of-strain
                                 double *f,          // distribution function
                                 double *feq,        // equilibrium distribution function
                                 double *f_new)      // new distribution function
{
    // compute the global "i" and "j" location handled by this thread

    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    // bound checking
    if( (i < 1) || (i > (N-2)) || (j < 1) || (j > (N-2)) ) return;

    // natural index
    const int index = i*N + j;  // column-major ordering

    // calculate fluid viscosity based on the Reynolds number
    double kinematicViscosity = LID_VELOCITY * (double) N / REYNOLDS_NUMBER;

    // calculate relaxation time tau
    double tau =  0.5 + 3.0 * kinematicViscosity;

    // collision
    for(int a=0;a<Q;a++) {
        int index_f = a + index*Q;
        double edotu = ex[a]*ux[index] + ey[a]*uy[index];
        double udotu = ux[index]*ux[index] + uy[index]*uy[index];
        feq[index_f] = rho[index] * wt[a] * (1 + 3*edotu + 4.5*edotu*edotu - 1.5*udotu);
    }

    // streaming from interior node points

    for(int a=0;a<Q;a++) {

        int index_f = a + index*Q;
        int index_nbr = (i+ex[a])*N + (j+ey[a]);
        int index_nbr_f = a + index_nbr * Q;
        int indexoppos = oppos[a] + index*Q;

        double tau_eff, tau_t, C_Smagorinsky;  // turbulence model parameters

        C_Smagorinsky = 0.16;

        // tau_t = additional contribution to the relaxation time 
        //         because of the "eddy viscosity" model
        // =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=
        // REFERENCE: Krafczyk M., Tolke J. and Luo L.-S. (2003)
        //            Large-Eddy Simulations with a Multiple-Relaxation-Time LBE Model
        //            International Journal of Modern Physics B, Vol.17, 33-39
        // =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=

        tau_t = 0.5*(pow(pow(tau,2) + 18.0*pow(C_Smagorinsky,2)*sigma[index],0.5) - tau);

        // the effective relaxation time accounts for the additional "eddy viscosity"
        // effects. Note that tau_eff now varies from point to point in the domain, and is
        // larger for large strain rates. If the strain rate is zero, tau_eff = 0 and we
        // revert back to the original (laminar) LBM scheme where tau_eff = tau.

        tau_eff = tau + tau_t;

        // post-collision distribution at (i,j) along "a"
        double f_plus = f[index_f] - (f[index_f] - feq[index_f])/tau_eff;

        int iS = i + ex[a]; int jS = j + ey[a];

        if((iS==0) || (iS==N-1) || (jS==0) || (jS==N-1)) {
            // bounce back
            double ubdote = ux[index_nbr]*ex[a] + uy[index_nbr]*ey[a];
            f_new[indexoppos] = f_plus - 6.0 * DENSITY * wt[a] * ubdote;
        }
        else {
            // stream to neighbor
            f_new[index_nbr_f] = f_plus;
        }
    }
}

__global__ void everythingElse( // READ-ONLY parameters (used by this function but not changed)
                                 const int N, const int Q, const double DENSITY, const double LID_VELOCITY, const double REYNOLDS_NUMBER,
                                 const double *ex,      // x-component of aection vector
                                 const double *ey,      // x-component of aection vector
                                 const double *wt,   // weight factor for each aection
                                 const int *oppos,        // anti (opposite) vector for each aection

                                 // READ + WRITE parameters (get updated in this function)

                                 double *rho,         // density
                                 double *ux,         // X-velocity
                                 double *uy,         // Y-velocity
                                 double *sigma,      // rate-of-strain
                                 double *f,          // distribution function
                                 double *feq,        // equilibrium distribution function
                                 double *f_new)      // new distribution function
{
    // compute the global "i" and "j" location of this thread

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // bound checking
    if( (i < 1) || (i > (N-2)) || (j < 1) || (j > (N-2)) ) return;

    // natural index
    const int index = i*N + j;  // column-major ordering

    // push f_new into f
    for(int a=0;a<Q;a++) {
        int index_f = a + index*Q;
        f[index_f] = f_new[index_f];
    }

    // update density at interior nodes
    rho[index]=0.0;
    for(int a=0;a<Q;a++) {
        int index_f = a + index*Q;
        rho[index] += f_new[index_f];
    }

    // update velocity at interior nodes
    double velx=0.0;
    double vely=0.0;
    for(int a=0;a<Q;a++) {
        int index_f = a + index*Q;
        velx += f_new[index_f]*ex[a];
        vely += f_new[index_f]*ey[a];
    }
    ux[index] = velx/rho[index];
    uy[index] = vely/rho[index];

    // update the rate-of-strain field
    double sum_xx = 0.0, sum_xy = 0.0, sum_xz = 0.0;
    double sum_yx = 0.0, sum_yy = 0.0, sum_yz = 0.0;
    double sum_zx = 0.0, sum_zy = 0.0, sum_zz = 0.0;
    for(int a=1; a<Q; a++)
    {
        int index_f = a + index*Q;

        sum_xx = sum_xx + (f_new[index_f] - feq[index_f])*ex[a]*ex[a];
        sum_xy = sum_xy + (f_new[index_f] - feq[index_f])*ex[a]*ey[a];
        sum_xz = 0.0;
        sum_yx = sum_xy;
        sum_yy = sum_yy + (f_new[index_f] - feq[index_f])*ey[a]*ey[a];
        sum_yz = 0.0;
        sum_zx = 0.0;
        sum_zy = 0.0;
        sum_zz = 0.0;
    }

    // evaluate |S| (magnitude of the strain-rate)
    sigma[index] = pow(sum_xx,2) + pow(sum_xy,2) + pow(sum_xz,2)
                 + pow(sum_yx,2) + pow(sum_yy,2) + pow(sum_yz,2)
                 + pow(sum_zx,2) + pow(sum_zy,2) + pow(sum_zz,2);

    sigma[index] = pow(sigma[index],0.5);
}

int main(int argc, char* argv[])
{
    try {

        // check whether to do graphics stuff or not
        bool isconsole = (argc == 2 && argv[1][0] == '-');

        // allocate memory on the GPU

        // the base vectors and associated weight coefficients (GPU)
        double *ex, *ey, *wt;  // pointers to device (GPU) memory
        hipMalloc((void **)&ex,Q*sizeof(double));
        hipMalloc((void **)&ey,Q*sizeof(double));
        hipMalloc((void **)&wt,Q*sizeof(double));

        // ant vector (GPU)
        int *oppos;  // gpu memory
        hipMalloc((void **)&oppos,Q*sizeof(int));

        // distribution functions
        double *f, *feq, *f_new;
        hipMalloc((void **)&f,N*N*Q*sizeof(double));
        hipMalloc((void **)&feq,N*N*Q*sizeof(double));
        hipMalloc((void **)&f_new,N*N*Q*sizeof(double));

        // density and velocity
        double *rho, *ux, *uy;
        hipMalloc((void **)&rho,N*N*sizeof(double));
        hipMalloc((void **)&ux,N*N*sizeof(double));
        hipMalloc((void **)&uy,N*N*sizeof(double));

        // rate-of-strain
        double *sigma;
        hipMalloc((void **)&sigma,N*N*sizeof(double));

        // assign a 2D distribution of CUDA "threads" within each CUDA "block"    
        int threadsAlongX=16, threadsAlongY=16;
        dim3 dimBlock(threadsAlongX, threadsAlongY, 1);

        // calculate number of blocks along X and Y in a 2D CUDA "grid"
        dim3 dimGrid( ceil(float(N)/float(dimBlock.x)), ceil(float(N)/float(dimBlock.y)), 1 );

        // launch GPU kernel to initialize all fields
        initialize<<<dimGrid,dimBlock>>>(N, Q, DENSITY, LID_VELOCITY,
                                         ex, ey, wt, oppos,
                                         rho, ux, uy, sigma,
                                         f, feq, f_new);

        // time integration
        int time=0;
        while(time<TIME_STEPS) {

            time++;

            std::cout << "Time = " << time << std::endl;

            collideAndStream<<<dimGrid,dimBlock >>>(N, Q, DENSITY, LID_VELOCITY, REYNOLDS_NUMBER,
                                                    ex, ey, wt, oppos,
                                                    rho, ux, uy, sigma,
                                                    f, feq, f_new);

            // collideAndStream and everythingElse were originally one kernel
            // they were separated out to make all threads synchronize globally
            // before moving on to the next set of calculations

            everythingElse<<<dimGrid,dimBlock >>>(N, Q, DENSITY, LID_VELOCITY, REYNOLDS_NUMBER,
                                                  ex, ey, wt, oppos,
                                                  rho, ux, uy, sigma,
                                                  f, feq, f_new);

            // this is where ArrayFire is currently used
            // the cool thing is you don't need to move the GPU arrays back to the
            // CPU for visualizing them. And of course, we have in-situ graphics

    //      double curl_min = 0, curl_max = 0;

            if (time % 10 == 0) {
                if(!isconsole) {
                    array U(N,N,ux,afDevice);
                    array V(N,N,uy,afDevice);
                    array umag = pow(U*U + V*V, 0.5);

//                  array dUdx,dUdy,dVdx,dVdy;
//                  grad(dUdx,dUdy,U);
//                  grad(dVdx,dVdy,V);
//                  array curl = dVdx - dUdy;

//                  double2 extrema = minmax<double2>(curl);
//                  std::cout << "Curl --- min " << extrema.x << "  max " << extrema.y << std::endl;

//                  if (extrema.x < curl_min) curl_min = extrema.x;
//                  if (extrema.y > curl_max) curl_max = extrema.y;

//                  curl(0) = -0.1;
//                  curl(N) = +0.1;
                    fig("color","heat");
                    image(umag);
                }
            }
    
        }

    } catch (af::exception& e) {
        fprintf(stderr, "%s\n", e.what());
        throw;
    }

    return 0;
}
