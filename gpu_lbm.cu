#include "hip/hip_runtime.h"
/*

Simulation of flow inside a 2D square cavity
using the lattice Boltzmann method (LBM)

Written by:       Abhijit Joshi (abhijit@accelereyes.com)

Last modified on: Thursday, July 18 2013 @12:08 pm

Build instructions: make (uses Makefile present in this folder)

Run instructions: optirun ./gpu_lbm

*/

#include<iostream>
#include<stdio.h>
#include<arrayfire.h>
using namespace af;

// problem parameters

const int     N = 128;                  // number of node points along X and Y (cavity length in lattice units)
const int     TIME_STEPS = 1000000;     // number of time steps for which the simulation is run
const double  REYNOLDS_NUMBER = 1E4;    // REYNOLDS_NUMBER = LID_VELOCITY * N / kinematicViscosity

// don't change these unless you know what you are doing

const int     Q = 9;                    // number of discrete velocity directions used
const double  DENSITY = 2.7;            // fluid density in lattice units
const double  LID_VELOCITY = 0.05;      // lid velocity in lattice units

// initialize values for direction vectors, density, velocity and distribution functions on the GPU

__global__ void initialize(const int N, const int Q, const double DENSITY, const double LID_VELOCITY, 
                           double *ex, double *ey, double *wt, int *oppos, 
                           double *rho, double *ux, double *uy, double* sigma, 
                           double *f, double *feq, double *f_new)
{
    // D2Q9 model base velocities and weights

    ex[0] =  0.0;   ey[0] =  0.0;   wt[0] = 4.0 /  9.0;
    ex[1] =  1.0;   ey[1] =  0.0;   wt[1] = 1.0 /  9.0;
    ex[2] =  0.0;   ey[2] =  1.0;   wt[2] = 1.0 /  9.0;
    ex[3] = -1.0;   ey[3] =  0.0;   wt[3] = 1.0 /  9.0;
    ex[4] =  0.0;   ey[4] = -1.0;   wt[4] = 1.0 /  9.0;
    ex[5] =  1.0;   ey[5] =  1.0;   wt[5] = 1.0 / 36.0;
    ex[6] = -1.0;   ey[6] =  1.0;   wt[6] = 1.0 / 36.0;
    ex[7] = -1.0;   ey[7] = -1.0;   wt[7] = 1.0 / 36.0;
    ex[8] =  1.0;   ey[8] = -1.0;   wt[8] = 1.0 / 36.0;

    // define opposite (anti) directions (useful for implementing bounce back)

    oppos[0] = 0;      //      6        2        5
    oppos[1] = 3;      //               ^
    oppos[2] = 4;      //               |
    oppos[3] = 1;      //               |
    oppos[4] = 2;      //      3 <----- 0 -----> 1
    oppos[5] = 7;      //               |
    oppos[6] = 8;      //               |
    oppos[7] = 5;      //               v
    oppos[8] = 6;      //      7        4        8

    // compute the global "i" and "j" location handled by this thread

    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    // natural index for location (i,j)

    int ixy = i*N+j;

    // initialize density and velocity fields inside the cavity

    rho[ixy] = DENSITY;
    ux[ixy] = 0.0;
    uy[ixy] = 0.0;
    sigma[ixy] = 0.0;

    // specify boundary condition for the moving lid

    if((i>0) && (i<N-1) &&(j==0)) ux[ixy] = LID_VELOCITY;

    // assign initial values for distribution functions
    // along various directions using equilibriu, functions

    for(int dir=0;dir<Q;dir++) {

        int index = i*N*Q + j*Q + dir;

        double edotu = ex[dir]*ux[ixy] + ey[dir]*uy[ixy];
        double udotu = ux[ixy]*ux[ixy] + uy[ixy]*uy[ixy];

        feq[index]   = rho[ixy] * wt[dir] * (1.0 + 3.0*edotu + 4.5*edotu*edotu - 1.5*udotu);
        f[index]     = feq[index];
        f_new[index] = feq[index];

    }
}

// this function updates the values of the distribution functions at all points along all directions
// carries out one lattice time-step (streaming + collision) in the algorithm

__global__ void collideAndStream( // READ-ONLY parameters (used by this function but not changed)
                                 const int N, const int Q, const double DENSITY, const double LID_VELOCITY, const double REYNOLDS_NUMBER,
                                 const double *ex,      // x-component of direction vector
                                 const double *ey,      // x-component of direction vector
                                 const double *wt,   // weight factor for each direction
                                 const int *oppos,        // anti (opposite) vector for each direction

                                 // READ + WRITE parameters (get updated in this function)

                                 double *rho,         // density
                                 double *ux,         // X-velocity
                                 double *uy,         // Y-velocity
                                 double *sigma,      // rate-of-strain
                                 double *f,          // distribution function
                                 double *feq,        // equilibrium distribution function
                                 double *f_new)      // new distribution function
{
    // calculate fluid viscosity based on the Reynolds number
    double kinematicViscosity = LID_VELOCITY * (double) N / REYNOLDS_NUMBER;


    // calculate relaxation time tau
    double tau =  0.5 + 3.0 * kinematicViscosity;

    // compute the global "i" and "j" location of this thread

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    {
        // collision
        if((i>0) && (i<N-1) && (j>0) && (j<N-1)) {
                int ixy = i*N + j;
                for(int dir=0;dir<Q;dir++) {
                    int index = i*N*Q + j*Q + dir;
                    double edotu = ex[dir]*ux[ixy] + ey[dir]*uy[ixy];
                    double udotu = ux[ixy]*ux[ixy] + uy[ixy]*uy[ixy];
                    feq[index] = rho[ixy] * wt[dir] * (1 + 3*edotu + 4.5*edotu*edotu - 1.5*udotu);
                }
        }

        // streaming from interior node points

        if((i>0) && (i<N-1) && (j>0) && (j<N-1)) {
                for(int dir=0;dir<Q;dir++) {

                    int index = i*N*Q + j*Q + dir;
                    int index_new = (i+ex[dir])*N*Q + (j+ey[dir])*Q + dir;
                    int indexoppos = i*N*Q + j*Q + oppos[dir];

                    double tau_eff, tau_t, C_Smagorinsky;  // turbulence model parameters

                    C_Smagorinsky = 0.16;

                    // tau_t = additional contribution to the relaxation time 
                    //         because of the "eddy viscosity" model
                    // =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=
                    // REFERENCE: Krafczyk M., Tolke J. and Luo L.-S. (2003)
                    //            Large-Eddy Simulations with a Multiple-Relaxation-Time LBE Model
                    //            International Journal of Modern Physics B, Vol.17, 33-39
                    // =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=

                    tau_t = 0.5*(pow(pow(tau,2) + 18.0*pow(C_Smagorinsky,2)*sigma[i*N+j],0.5) - tau);

//                  if((i==250) && (j==1)) printf("tau_t = %f\n",tau_t);

                    // the effective relaxation time accounts for the additional "eddy viscosity"
                    // effects. Note that tau_eff now varies from point to point in the domain, and is
                    // larger for large strain rates. If the strain rate is zero, tau_eff = 0 and we
                    // revert back to the original (laminar) LBM scheme where tau_eff = tau.

                    tau_eff = tau + tau_t;

                    // post-collision distribution at (i,j) along "dir"
                    double f_plus = f[index] - (f[index] - feq[index])/tau_eff;

//                  if ((i==250) && (j==1)) printf("j = %d dir = %d   f_plus = %f\n",j,dir,f_plus);

                    int iS = i + ex[dir]; int jS = j + ey[dir];

                    if((iS==0) || (iS==N-1) || (jS==0) || (jS==N-1)) {
                        // bounce back
                        int ixy_nbr = iS*N + jS;
                        double ubdote = ux[ixy_nbr]*ex[dir] + uy[ixy_nbr]*ey[dir];
                        f_new[indexoppos] = f_plus - 6.0 * DENSITY * wt[dir] * ubdote;

//                      if((i==250) && (j+ey[dir]==0)) {
//                        printf("ubdote = %f  Bounce backed f(%d) = %f\n",ubdote, oppos[dir],f_new[indexoppos]);
//                      }


                    }
                    else {
                        // stream to neighbor
                        f_new[index_new] = f_plus;
                    }
                }
        }
    }
}

__global__ void everythingElse( // READ-ONLY parameters (used by this function but not changed)
                                 const int N, const int Q, const double DENSITY, const double LID_VELOCITY, const double REYNOLDS_NUMBER,
                                 const double *ex,      // x-component of direction vector
                                 const double *ey,      // x-component of direction vector
                                 const double *wt,   // weight factor for each direction
                                 const int *oppos,        // anti (opposite) vector for each direction

                                 // READ + WRITE parameters (get updated in this function)

                                 double *rho,         // density
                                 double *ux,         // X-velocity
                                 double *uy,         // Y-velocity
                                 double *sigma,      // rate-of-strain
                                 double *f,          // distribution function
                                 double *feq,        // equilibrium distribution function
                                 double *f_new)      // new distribution function
{
    // compute the global "i" and "j" location of this thread

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    {
        // push f_new into f
        if((i>0) && (i<N-1) && (j>0) && (j<N-1)) {
                for(int dir=0;dir<Q;dir++) {
                    int index = i*N*Q + j*Q + dir;
                    f[index] = f_new[index];
//                  if ((i==250) && (j==1)) printf("j = %d dir = %d   f_new = %f\n",j,dir,f_new[index]);
                }
        }

        // update density at interior nodes
        if((i>0) && (i<N-1) && (j>0) && (j<N-1)) {
                rho[i*N+j]=0.0;
                for(int dir=0;dir<Q;dir++) {
                    int index = i*N*Q + j*Q + dir;
                    rho[i*N+j] += f_new[index];
//                  if ((i==250) && (j==1)) printf("f(%d) = %f\n",dir,f_new[index]);
                }
        }

        // update velocity at interior nodes
        if((i>0) && (i<N-1) && (j>0) && (j<N-1)) {
                double velx=0.0;
                double vely=0.0;
                for(int dir=0;dir<Q;dir++) {
                    int index = i*N*Q + j*Q + dir;
                    velx+=f_new[index]*ex[dir];
                    vely+=f_new[index]*ey[dir];
                }
                int ixy = i*N+j;
                ux[ixy] = velx/rho[ixy] ;
                uy[ixy] = vely/rho[ixy] ;
        }

        // update the rate-of-strain field
        if((i>0) && (i<N-1) && (j>0) && (j<N-1)) {
            double sum_xx = 0.0, sum_xy = 0.0, sum_xz = 0.0;
            double sum_yx = 0.0, sum_yy = 0.0, sum_yz = 0.0;
            double sum_zx = 0.0, sum_zy = 0.0, sum_zz = 0.0;
            for(int dir=1; dir<Q; dir++)
            {
              int index = i*N*Q + j*Q + dir;

              sum_xx = sum_xx + (f_new[index] - feq[index])*ex[dir]*ex[dir];
              sum_xy = sum_xy + (f_new[index] - feq[index])*ex[dir]*ey[dir];
              sum_xz = 0.0;
              sum_yx = sum_xy;
              sum_yy = sum_yy + (f_new[index] - feq[index])*ey[dir]*ey[dir];
              sum_yz = 0.0;
              sum_zx = 0.0;
              sum_zy = 0.0;
              sum_zz = 0.0;
            }

            // evaluate |S| (magnitude of the strain-rate)
            int i_j    = i*N+j;

            sigma[i_j] = pow(sum_xx,2) + pow(sum_xy,2) + pow(sum_xz,2)
                        + pow(sum_yx,2) + pow(sum_yy,2) + pow(sum_yz,2)
                        + pow(sum_zx,2) + pow(sum_zy,2) + pow(sum_zz,2);

            sigma[i_j] = pow(sigma[i_j],0.5);

        }

    }
}

int main(int argc, char* argv[])
{
    try {

        // check whether to do graphics stuff or not
        bool isconsole = (argc == 2 && argv[1][0] == '-');

        // allocate memory on the GPU

        // the base vectors and associated weight coefficients (GPU)
        double *ex, *ey, *wt;  // pointers to device (GPU) memory
        hipMalloc((void **)&ex,Q*sizeof(double));
        hipMalloc((void **)&ey,Q*sizeof(double));
        hipMalloc((void **)&wt,Q*sizeof(double));

        // ant vector (GPU)
        int *oppos;  // gpu memory
        hipMalloc((void **)&oppos,Q*sizeof(int));

        // distribution functions
        double *f, *feq, *f_new;
        hipMalloc((void **)&f,N*N*Q*sizeof(double));
        hipMalloc((void **)&feq,N*N*Q*sizeof(double));
        hipMalloc((void **)&f_new,N*N*Q*sizeof(double));

        // density and velocity
        double *rho, *ux, *uy;
        hipMalloc((void **)&rho,N*N*sizeof(double));
        hipMalloc((void **)&ux,N*N*sizeof(double));
        hipMalloc((void **)&uy,N*N*sizeof(double));

        // rate-of-strain
        double *sigma;
        hipMalloc((void **)&sigma,N*N*sizeof(double));

        // assign a 2D distribution of CUDA "threads" within each CUDA "block"    
        int threadsAlongX=16, threadsAlongY=16;
        dim3 dimBlock(threadsAlongX, threadsAlongY, 1);

        // calculate number of blocks along X and Y in a 2D CUDA "grid"
        dim3 dimGrid( ceil(float(N)/float(dimBlock.x)), ceil(float(N)/float(dimBlock.y)), 1 );

        // launch GPU kernel to initialize all fields
        initialize<<<dimGrid,dimBlock>>>(N, Q, DENSITY, LID_VELOCITY,
                                         ex, ey, wt, oppos,
                                         rho, ux, uy, sigma,
                                         f, feq, f_new);

        // time integration
        int time=0;
        while(time<TIME_STEPS) {

            time++;

            std::cout << "Time = " << time << std::endl;

            collideAndStream<<<dimGrid,dimBlock >>>(N, Q, DENSITY, LID_VELOCITY, REYNOLDS_NUMBER,
                                                    ex, ey, wt, oppos,
                                                    rho, ux, uy, sigma,
                                                    f, feq, f_new);

            // collideAndStream and everythingElse were originally one kernel
            // they were separated out to make all threads synchronize globally
            // before moving on to the next set of calculations

            everythingElse<<<dimGrid,dimBlock >>>(N, Q, DENSITY, LID_VELOCITY, REYNOLDS_NUMBER,
                                                  ex, ey, wt, oppos,
                                                  rho, ux, uy, sigma,
                                                  f, feq, f_new);

            // this is where ArrayFire is currently used
            // the cool thing is you don't need to move the GPU arrays back to the
            // CPU for visualizing them. And of course, we have in-situ graphics

    //      double curl_min = 0, curl_max = 0;

            if (time % 10 == 0) {
                if(!isconsole) {
                    array U(N,N,ux,afDevice);
                    array V(N,N,uy,afDevice);
                    array umag = pow(U*U + V*V, 0.5);

//                  array dUdx,dUdy,dVdx,dVdy;
//                  grad(dUdx,dUdy,U);
//                  grad(dVdx,dVdy,V);
//                  array curl = dVdx - dUdy;

//                  double2 extrema = minmax<double2>(curl);
//                  std::cout << "Curl --- min " << extrema.x << "  max " << extrema.y << std::endl;

//                  if (extrema.x < curl_min) curl_min = extrema.x;
//                  if (extrema.y > curl_max) curl_max = extrema.y;

//                  curl(0) = -0.1;
//                  curl(N) = +0.1;
                    fig("color","heat");
                    image(umag);
                }
            }
    
        }

    } catch (af::exception& e) {
        fprintf(stderr, "%s\n", e.what());
        throw;
    }

    return 0;
}
